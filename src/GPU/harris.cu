#include "hip/hip_runtime.h"
#include <png.h>
#include <spdlog/spdlog.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>

#include <cassert>
#include <cstdio>

#include "harris.hpp"

#define RGBA_DIM 4

[[gnu::noinline]] void _abortError(const char *msg, const char *fname,
                                   int line) {
  hipError_t err = hipGetLastError();
  spdlog::error("{} ({}, line: {})", msg, fname, line);
  spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
  std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

__constant__ float gauss_kernel[7][7] = {
    {1.23409804e-04, 1.50343919e-03, 6.73794700e-03, 1.11089965e-02,
     6.73794700e-03, 1.50343919e-03, 1.23409804e-04},
    {1.50343919e-03, 1.83156389e-02, 8.20849986e-02, 1.35335283e-01,
     8.20849986e-02, 1.83156389e-02, 1.50343919e-03},
    {6.73794700e-03, 8.20849986e-02, 3.67879441e-01, 6.06530660e-01,
     3.67879441e-01, 8.20849986e-02, 6.73794700e-03},
    {1.11089965e-02, 1.35335283e-01, 6.06530660e-01, 1.00000000e+00,
     6.06530660e-01, 1.35335283e-01, 1.11089965e-02},
    {6.73794700e-03, 8.20849986e-02, 3.67879441e-01, 6.06530660e-01,
     3.67879441e-01, 8.20849986e-02, 6.73794700e-03},
    {1.50343919e-03, 1.83156389e-02, 8.20849986e-02, 1.35335283e-01,
     8.20849986e-02, 1.83156389e-02, 1.50343919e-03},
    {1.23409804e-04, 1.50343919e-03, 6.73794700e-03, 1.11089965e-02,
     6.73794700e-03, 1.50343919e-03, 1.23409804e-04}};

__constant__ float gauss_derivative_x[7][7] = {
    {3.70229412e-04, 3.00687839e-03, 6.73794700e-03, 0.00000000e+00,
     -6.73794700e-03, -3.00687839e-03, -3.70229412e-04},
    {4.51031758e-03, 3.66312778e-02, 8.20849986e-02, 0.00000000e+00,
     -8.20849986e-02, -3.66312778e-02, -4.51031758e-03},
    {2.02138410e-02, 1.64169997e-01, 3.67879441e-01, 0.00000000e+00,
     -3.67879441e-01, -1.64169997e-01, -2.02138410e-02},
    {3.33269896e-02, 2.70670566e-01, 6.06530660e-01, 0.00000000e+00,
     -6.06530660e-01, -2.70670566e-01, -3.33269896e-02},
    {2.02138410e-02, 1.64169997e-01, 3.67879441e-01, 0.00000000e+00,
     -3.67879441e-01, -1.64169997e-01, -2.02138410e-02},
    {4.51031758e-03, 3.66312778e-02, 8.20849986e-02, 0.00000000e+00,
     -8.20849986e-02, -3.66312778e-02, -4.51031758e-03},
    {3.70229412e-04, 3.00687839e-03, 6.73794700e-03, 0.00000000e+00,
     -6.73794700e-03, -3.00687839e-03, -3.70229412e-04}};

__constant__ float gauss_derivative_y[7][7] = {
    {3.70229412e-04, 4.51031758e-03, 2.02138410e-02, 3.33269896e-02,
     2.02138410e-02, 4.51031758e-03, 3.70229412e-04},
    {3.00687839e-03, 3.66312778e-02, 1.64169997e-01, 2.70670566e-01,
     1.64169997e-01, 3.66312778e-02, 3.00687839e-03},
    {6.73794700e-03, 8.20849986e-02, 3.67879441e-01, 6.06530660e-01,
     3.67879441e-01, 8.20849986e-02, 6.73794700e-03},
    {0.00000000e+00, 0.00000000e+00, 0.00000000e+00, 0.00000000e+00,
     0.00000000e+00, 0.00000000e+00, 0.00000000e+00},
    {-6.73794700e-03, -8.20849986e-02, -3.67879441e-01, -6.06530660e-01,
     -3.67879441e-01, -8.20849986e-02, -6.73794700e-03},
    {-3.00687839e-03, -3.66312778e-02, -1.64169997e-01, -2.70670566e-01,
     -1.64169997e-01, -3.66312778e-02, -3.00687839e-03},
    {-3.70229412e-04, -4.51031758e-03, -2.02138410e-02, -3.33269896e-02,
     -2.02138410e-02, -4.51031758e-03, -3.70229412e-04}};

__constant__ unsigned char structElement[25][25] = {
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0},
    {0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0},
    {0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0},
    {0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0},
    {0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0},
    {0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0},
    {0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0},
    {0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0},
    {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
    {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
    {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
    {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
    {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
    {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
    {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
    {0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0},
    {0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0},
    {0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0},
    {0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0},
    {0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0},
    {0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0},
    {0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0},
    {0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0}};

// Device code
__global__ void rgba2gray(unsigned char *rgba_buffer, float *gray_buffer,
                          int width, int height, size_t rgba_pitch,
                          size_t gray_pitch) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height) return;

  int y_rgba = y * rgba_pitch;

  float *line = (float *)((char *)gray_buffer + y * gray_pitch);
  line[x] = 0.299 * rgba_buffer[y_rgba + x * 4] +
            0.587 * rgba_buffer[y_rgba + x * 4 + 1] +
            0.114 * rgba_buffer[y_rgba + x * 4 + 2];
}

__global__ void computeDerivatives(float *image, int width, int height,
                                   size_t pitch, float *imx2, float *imxy,
                                   float *imy2) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height) return;

  float imx = 0;
  float imy = 0;

  for (int i = 0; i < 7; i++) {
    for (int j = 0; j < 7; j++) {
      int tmpY = min(max(0, y + i - 3), height - 1);
      int tmpX = min(max(0, x + j - 3), width - 1);

      float *line = (float *)((char *)image + tmpY * pitch);
      imx += gauss_derivative_x[6 - i][6 - j] * line[tmpX];
      imy += gauss_derivative_y[6 - i][6 - j] * line[tmpX];
    }
  }

  ((float *)((char *)imx2 + y * pitch))[x] = imx * imx;
  ((float *)((char *)imxy + y * pitch))[x] = imx * imy;
  ((float *)((char *)imy2 + y * pitch))[x] = imy * imy;
}

__global__ void computeHarrisResponse(int width, int height, size_t pitch,
                                      float *imx2, float *imxy, float *imy2,
                                      float *response) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height) return;

  float Wxx = 0;
  float Wxy = 0;
  float Wyy = 0;

  for (int i = 0; i < 7; i++) {
    for (int j = 0; j < 7; j++) {
      int tmpY = min(max(0, y + i - 3), height - 1);
      int tmpX = min(max(0, x + j - 3), width - 1);

      Wxx += gauss_kernel[6 - i][6 - j] *
             ((float *)((char *)imx2 + tmpY * pitch))[tmpX];
      Wxy += gauss_kernel[6 - i][6 - j] *
             ((float *)((char *)imxy + tmpY * pitch))[tmpX];
      Wyy += gauss_kernel[6 - i][6 - j] *
             ((float *)((char *)imy2 + tmpY * pitch))[tmpX];
    }
  }

  float WxxWyy = Wxx * Wyy;
  float Wxy2 = Wxy * Wxy;

  float Wdet = WxxWyy - Wxy2;
  float WtrEps = Wxx + Wyy + 1;

  ((float *)((char *)response + y * pitch))[x] = Wdet / WtrEps;
}

__global__ void morphoDilate(float *input, int width, int height, size_t pitch,
                             float *output) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height) return;

  float pixel = 0;

  for (int i = 0; i < 25; i++) {
    int testedY = y + i - 12;
    for (int j = 0; j < 25; j++) {
      int testedX = x + j - 12;
      float value = ((float *)((char *)input + testedY * pitch))[testedX];
      if (testedY >= 0 && testedY < height && testedX >= 0 && testedX < width &&
          structElement[i][j] && value > pixel)
        pixel = value;
    }
  }

  ((float *)((char *)output + y * pitch))[x] = pixel;
}

__global__ void removePadding(float *inputBuffer, int width, int height,
                              size_t pitch, float *outputBuffer) {
  int line = blockDim.x * blockIdx.x + threadIdx.x;
  if (line < height)
    memcpy(outputBuffer + line * width, (char *)inputBuffer + line * pitch,
           width * sizeof(float));
}

__global__ void harrisThreshold(float *harris, int width, int height,
                                float ref) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= width * height) return;

  float val = harris[i];
  harris[i] = val > ref ? val : 0;
}

struct PointCmp {
  __host__ __device__ bool operator()(float3 a, float3 b) { return a.x > b.x; }
};

__global__ void retrieveKeypoints(float *harris, float *dilatedHarris,
                                  int width, int height, size_t pitch,
                                  float3 *keypoints) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= width * height) return;

  int y = i / width;
  int x = i % width;

  float val = harris[i];
  float dilatedVal = ((float *)((char *)dilatedHarris + y * pitch))[x];
  float delta = abs(dilatedVal - val);

  float coef = val && (delta < __FLT_EPSILON__) ? 1 : 0;
  keypoints[i] = make_float3(harris[i] * coef, x, y);
}

float *detectHarrisPointsGPU(unsigned char **rgba_image, int width, int height,
                             size_t max_keypoints, float threshold,
                             size_t *nbFound) {
  hipError_t rc = hipSuccess;

  // Allocate device memory
  unsigned char *rgba_buffer;
  size_t rgba_pitch;
  rc = hipMallocPitch(&rgba_buffer, &rgba_pitch,
                       width * RGBA_DIM * sizeof(unsigned char), height);
  if (rc) abortError("Fail buffer allocation");

  float *gray_buffer;
  size_t pitch;
  rc = hipMallocPitch(&gray_buffer, &pitch, width * sizeof(float), height);
  if (rc) abortError("Fail buffer allocation");

  float *imx2;
  rc = hipMallocPitch(&imx2, &pitch, width * sizeof(float), height);
  if (rc) abortError("Fail buffer allocation");

  float *imxy;
  rc = hipMallocPitch(&imxy, &pitch, width * sizeof(float), height);
  if (rc) abortError("Fail buffer allocation");

  float *imy2;
  rc = hipMallocPitch(&imy2, &pitch, width * sizeof(float), height);
  if (rc) abortError("Fail buffer allocation");

  float *response_flat;
  rc = hipMalloc(&response_flat, width * height * sizeof(float));
  if (rc) abortError("Fail buffer allocation");

  float3 *keypoints;
  rc = hipMalloc(&keypoints, width * height * sizeof(float3));
  if (rc) abortError("Fail buffer allocation");

  // Copy image to GPU memory
  rc = hipMemcpy2D(rgba_buffer, rgba_pitch, *rgba_image, width * RGBA_DIM,
                    width * RGBA_DIM * sizeof(unsigned char), height,
                    hipMemcpyHostToDevice);
  if (rc) abortError("Unable to copy image to GPU memory");

  // Run the kernel with blocks of size 64 x 64
  int bsize = 32;
  int w = std::ceil((float)width / bsize);
  int h = std::ceil((float)height / bsize);

  spdlog::debug("running kernel of size ({},{})", w, h);

  dim3 dimBlock(bsize, bsize);
  dim3 dimGrid(w, h);
  rgba2gray<<<dimGrid, dimBlock>>>(rgba_buffer, gray_buffer, width, height,
                                   rgba_pitch, pitch);

  computeDerivatives<<<dimGrid, dimBlock>>>(gray_buffer, width, height, pitch,
                                            imx2, imxy, imy2);
  if (hipPeekAtLastError()) abortError("Computation Error");

  float *response = gray_buffer;

  computeHarrisResponse<<<dimGrid, dimBlock>>>(width, height, pitch, imx2, imxy,
                                               imy2, response);
  if (hipPeekAtLastError()) abortError("Computation Error");

  removePadding<<<(int)std::ceil(height / 1024.0), 1024>>>(
      response, width, height, pitch, response_flat);
  if (hipPeekAtLastError()) abortError("Computation Error");

  thrust::device_vector<float> vector(response_flat,
                                      response_flat + (width * height));

  thrust::pair minmax = thrust::minmax_element(vector.begin(), vector.end());

  float min = *(minmax.first);
  float max = *(minmax.second);

  float ref = min + threshold * (max - min);

  harrisThreshold<<<std::ceil(width * height / 1024.0), 1024>>>(
      response_flat, width, height, ref);
  if (hipPeekAtLastError()) abortError("Computation Error");

  float *dilated = imx2;

  morphoDilate<<<dimGrid, dimBlock>>>(response, width, height, pitch, dilated);

  if (hipPeekAtLastError()) abortError("Computation Error");

  retrieveKeypoints<<<std::ceil(width * height / 1024.0), 1024>>>(
      response_flat, dilated, width, height, pitch, keypoints);
  if (hipPeekAtLastError()) abortError("Computation Error");

  thrust::device_vector<float3> keypoints_vec(keypoints,
                                              keypoints + (width * height));

  thrust::sort(keypoints_vec.begin(), keypoints_vec.end(), PointCmp());

  size_t limit = 0;
  while (limit < keypoints_vec.size() && limit < max_keypoints) {
    if (((float3)keypoints_vec[limit]).x == 0)
      break;

    limit++;
  }

  float *result;
  rc = hipHostMalloc(&result, width * height * sizeof(float3));
  if (rc) abortError("Fail buffer allocation");

  rc = hipMemcpy(result, keypoints_vec.data().get(), limit * sizeof(float3),
                  hipMemcpyDeviceToHost);
  if (rc) abortError("Fail device to host copy");
  *nbFound = limit;

  // Free
  rc = hipFree(rgba_buffer);
  if (rc) abortError("Unable to free memory");

  rc = hipFree(gray_buffer);
  if (rc) abortError("Unable to free memory");

  rc = hipFree(imx2);
  if (rc) abortError("Unable to free memory");

  rc = hipFree(imxy);
  if (rc) abortError("Unable to free memory");

  rc = hipFree(imy2);
  if (rc) abortError("Unable to free memory");

  rc = hipFree(response_flat);
  if (rc) abortError("Unable to free memory");

  rc = hipFree(keypoints);
  if (rc) abortError("Unable to free memory");

  return result;
}
